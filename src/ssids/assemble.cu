#include "hip/hip_runtime.h"
#ifdef __cplusplus
#include <cmath>
#else
#include <math.h>
#endif

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>

#include "spral_cuda_cuda_check.h"
#include "ssids_gpu_kernels_datatypes.h"

#ifdef SPRAL_SINGLE
#define precision_ float
#define load_nodes_type load_nodes_type_single
#define assemble_cp_type assemble_cp_type_single
#define assemble_blk_type assemble_blk_type_single
#define assemble_delay_type assemble_delay_type_single
#define assemble assemble_single
#define add_delays add_delays_single
#define cu_load_nodes cu_load_nodes_single
#define cu_load_nodes_sc cu_load_nodes_sc_single
#define cu_max_abs cu_max_abs_single
#define spral_ssids_add_delays spral_ssids_add_delays_single
#define spral_ssids_assemble spral_ssids_assemble_single
#define spral_ssids_load_nodes spral_ssids_load_nodes_single
#define spral_ssids_load_nodes_sc spral_ssids_load_nodes_sc_single
#define spral_ssids_max_abs spral_ssids_max_abs_single
#else
#define precision_ double
#define load_nodes_type load_nodes_type_single
#define assemble_cp_type assemble_cp_type_double
#define assemble_blk_type assemble_blk_type_double
#define assemble_delay_type assemble_delay_type_double
#define assemble assemble_double
#define add_delays add_delays_double
#define cu_load_nodes cu_load_nodes_double
#define cu_load_nodes_sc cu_load_nodes_sc_double
#define cu_max_abs cu_max_abs_double
#define spral_ssids_add_delays spral_ssids_add_delays_double
#define spral_ssids_assemble spral_ssids_assemble_double
#define spral_ssids_load_nodes spral_ssids_load_nodes_double
#define spral_ssids_load_nodes_sc spral_ssids_load_nodes_sc_double
#define spral_ssids_max_abs spral_ssids_max_abs_double
#endif

#define HOGG_ASSEMBLE_TX 128  // Block height
#define HOGG_ASSEMBLE_TY 8    // Block width
#define HOGG_ASSEMBLE_NTX 32  // Number of threads x
#define HOGG_ASSEMBLE_NTY 4   // Number of threads y
#define ADD_DELAYS_TX 32
#define ADD_DELAYS_TY 4

namespace /* anon */ {

struct load_nodes_type {
  long nnz;    // Number of entries to map
  int lda;    // Leading dimension of A
  int ldl;    // Leading dimension of L
  precision_ *lcol; // Pointer to non-delay part of L
  long offn;   // Offset into nlist
  long offr;  // Offset into rlist
};

/*
 * Perform assembly according to nlist:
 * lval( nlist(2,i) ) = val( nlist(1,i) )    (in Fortran)
 *
 * Each block handles one node (regardless of size!!!)
 * Note: modified value lval is passed in via pointer in lndata, not as argument
 */
__global__ void
cu_load_nodes(
    const struct load_nodes_type *lndata,
    const long *nlist,
    const precision_ *aval
) {
   lndata += blockIdx.x;
   const long nnz = lndata->nnz;
   const int lda = lndata->lda;
   const int ldl = lndata->ldl;

   nlist += 2*lndata->offn;
   precision_ *const lval = lndata->lcol;
  
   for (int i = threadIdx.x; i < nnz; i += blockDim.x) {
     // Note: nlist is 1-indexed, not 0 indexed, so we have to adjust
     const int r = (nlist[2*i+1] - 1) % lda; // row index
     const int c = (nlist[2*i+1] - 1) / lda; // col index
     const long sidx = nlist[2*i+0] - 1; // source index
     lval[r + c*ldl] = aval[sidx];
   }
}

/*
 * Perform assembly according to nlist:
 * lval( nlist(2,i) ) = val( nlist(1,i) )    (in Fortran)
 * with the added twist of needing to perform a scaling at the same time
 *
 * Each block handles one node (regardless of size!!!)
 * Note: modified value lval is passed in via pointer in lndata, not as argument
 */
__global__ void
cu_load_nodes_sc(
    const struct load_nodes_type *lndata,
    const long *nlist,
    const int *rlist,
    const precision_ *scale,
    const precision_ *aval
) {
   lndata += blockIdx.x;
   const int nnz = lndata->nnz;
   const int lda = lndata->lda;
   const int ldl = lndata->ldl;

   nlist += 2*lndata->offn;
   precision_ *const lval = lndata->lcol;
   rlist += lndata->offr;
  
   for (int i = threadIdx.x; i < nnz; i += blockDim.x) {
      // Note: nlist and rlist are 1-indexed, not 0 indexed, so we adjust
      const int r = (nlist[2*i+1] - 1) % lda; // row index
      const int c = (nlist[2*i+1] - 1) / lda; // col index
      const long sidx = nlist[2*i+0] - 1; // source index
      const precision_ rs = scale[rlist[r] - 1]; // row scaling
      const precision_ cs = scale[rlist[c] - 1]; // col scaling
      lval[r + c*ldl] = rs * aval[sidx] * cs;
   }
}

// BLOCK_SIZE = blockDim.x
// maxabs must be initialized to zeros
template< typename ELEMENT_TYPE, unsigned int BLOCK_SIZE >
__global__ void
cu_max_abs( const long n, const ELEMENT_TYPE *const u, ELEMENT_TYPE *const maxabs )
{
  __shared__ volatile ELEMENT_TYPE tmax[BLOCK_SIZE];
  
  tmax[threadIdx.x] = 0.0;
  for ( long i = threadIdx.x + blockDim.x*blockIdx.x; i < n; 
        i += blockDim.x*gridDim.x ) {
    const ELEMENT_TYPE v = fabs(u[i]);
    if ( v > tmax[threadIdx.x] )
      tmax[threadIdx.x] = v;
  }
  __syncthreads();
  
  for ( int inc = 1; inc < BLOCK_SIZE; inc *= 2 ) {
    if ( 2*inc*threadIdx.x + inc < BLOCK_SIZE
        && tmax[2*inc*threadIdx.x + inc] > tmax[2*inc*threadIdx.x] )
      tmax[2*inc*threadIdx.x] = tmax[2*inc*threadIdx.x + inc];
    __syncthreads();
  }
  if ( threadIdx.x == 0 && tmax[0] > 0.0 )
    maxabs[blockIdx.x] = tmax[0];
}


/* Following data type describes a single child-parent assembly */
struct assemble_cp_type {
  // Parent data
  int pvoffset; // Offset to start of parent node values
  precision_ *pval; // Pointer to non-delay part of parent L
  int ldp; // Leading dimension of parent

  // Child data
  int cm; // Number of rows in child
  int cn; // Number of columns in child
  int ldc; // Leading dimension of child
  long cvoffset; // Offset to start of child node values
  precision_ *cv; // Pointer to start of child node values

  // Alignment data
  int *rlist_direct; // Pointer to start of child's rlist
  int *ind; // Pointer to start of child's contribution index

  // Sync data
  int sync_offset; // we watch sync[sync_offset]
  int sync_wait_for; // and wait for it to have value >= sync_wait_for
};

/* Following data type describes actions of single CUDA block */
struct assemble_blk_type {
  int cp; // node we're assembling into
  int blk; // block number of that node
};

/* Used to force volatile load of a declared non-volatile variable */
template <typename T_ELEM>
__inline__ __device__ T_ELEM loadVolatile(volatile T_ELEM *const vptr) {
  return *vptr;
}

/* Performs sparse assembly of a m x n child into a parent as dictated by
 * rlist_direct (supplied as part of cpdata).
 *
 * A lookup is performed in blkdata to determine which child-parent assembly
 * is to be performed next, and which block of that assembly this is.
 *
 * next_blk is used to ensure all blocks run in exact desired order.
 * sync[] is used to ensure dependencies are completed in the correct order.
 */
template <unsigned int blk_sz_x, unsigned int blk_sz_y,
          unsigned int ntx, unsigned nty>
void __global__ assemble(
    const struct assemble_blk_type *blkdata, // block mapping
    const struct assemble_cp_type *cpdata, // child-parent data
    const precision_ *const children, // pointer to array containing children
    precision_ *const parents, // pointer to array containing parents
    unsigned int *const next_blk, // gmem location used to determine next block
    volatile unsigned int *const sync // sync[cp] is #blocks completed so far for cp
) {
   // Get block number
   __shared__ volatile unsigned int mynext_blk;
   if(threadIdx.x==0 && threadIdx.y==0)
      mynext_blk = atomicAdd(next_blk, 1);
   __syncthreads();

   // Determine global information
   blkdata += mynext_blk;
   cpdata += blkdata->cp;
   int blk = blkdata->blk;
   int nx = (cpdata->cm-1) / blk_sz_x + 1; // number of blocks high child is
   int bx = blk % nx; // coordinate of block in x direction
   int by = blk / nx; // coordinate of block in y direction
   int ldc = cpdata->ldc;
   int ldp = cpdata->ldp;

   // Initialize local information
   int m = min(blk_sz_x, cpdata->cm - bx*blk_sz_x);
   int n = min(blk_sz_y, cpdata->cn - by*blk_sz_y);
   const precision_ *src = 
      cpdata->cv + ldc*by*blk_sz_y + bx*blk_sz_x;
   precision_ *dest = cpdata->pval;
   int *rows = cpdata->rlist_direct + bx*blk_sz_x;
   int *cols = cpdata->rlist_direct + by*blk_sz_y;

   // Wait for previous child of this parent to complete
   if(threadIdx.x==0 && threadIdx.y==0) {
      while(sync[cpdata->sync_offset] < cpdata->sync_wait_for) /**/;
   }
   __syncthreads();

   // Perform assembly
   for(int j=0; j<blk_sz_y/nty; j++) {
      if( threadIdx.y+j*nty < n ) {
         int col = cols[threadIdx.y+j*nty]-1;
         for(int i=0; i<blk_sz_x/ntx; i++) {
            if( threadIdx.x+i*ntx < m ) {
               int row = rows[threadIdx.x+i*ntx]-1;
               dest[row + col*ldp] += 
                  src[threadIdx.x+i*ntx + (threadIdx.y+j*nty)*ldc];
            }
         }
      }
   }

   // Record that we're done
   __syncthreads();
   if(threadIdx.x==0 && threadIdx.y==0) {
      atomicAdd((int*)&(sync[blkdata->cp]), 1);
   }
}

struct assemble_delay_type {
  int dskip; // Number of rows to skip for delays from later children
  int m; // Number of rows in child to copy
  int n; // Number of cols in child to copy
  int ldd; // Leading dimension of dest (parent)
  int lds; // Leading dimension of src (child)
  precision_ *dval; // Pointer to dest (parent)
  precision_ *sval; // Pointer to src (child)
  long roffset; // Offset to rlist_direct
};

/* Copies delays from child to parent using one block per parent 
 * Note: src and dest pointers both contained in dinfo
 */
void __global__ add_delays(
    struct assemble_delay_type *dinfo, // information on each block
    const int *rlist_direct // children's rows indices in parents
) {
   dinfo += blockIdx.x;
   const int dskip = dinfo->dskip; // number of delays
   const int m = dinfo->m; // number of rows
   const int n = dinfo->n; // number of cols
   const int ldd = dinfo->ldd; // leading dimension of dest
   const int lds = dinfo->lds; // leading dimension of src

   precision_ *const dest = dinfo->dval;
   const precision_ *const src = dinfo->sval;
   rlist_direct += dinfo->roffset;

   for ( int y = threadIdx.y; y < n; y += blockDim.y ) {
      for ( int x = threadIdx.x; x < m; x += blockDim.x ) {
         if ( x < n ) {
            dest[x + y*ldd] = src[x + y*lds];
         }
         else {
            int xt = dskip + rlist_direct[x - n] - 1;
            dest[xt + y*ldd] = src[x + y*lds];
         }
      }
   }
} 

} /* anon namespace */

/*******************************************************************************
 * Following routines are exported with C binding so can be called from Fortran
 ******************************************************************************/

extern "C" {

/* Invokes the add_delays<<<>>>() kernel */
void spral_ssids_add_delays( const hipStream_t *stream, int ndblk,
      struct assemble_delay_type *gpu_dinfo, int *rlist_direct ) {
   if ( ndblk == 0 ) return; // Nothing to see here
   dim3 threads(ADD_DELAYS_TX, ADD_DELAYS_TY);
   for ( int i = 0; i < ndblk; i += MAX_CUDA_BLOCKS ) {
      int nb = min(MAX_CUDA_BLOCKS, ndblk - i);
      add_delays
         <<< nb, threads, 0, *stream >>>
         ( gpu_dinfo + i, rlist_direct );
      CudaCheckError();
   }
}

/* Runs the kernel assemble<<<>>>() after setting up memory correctly. */
/* Requires gpu_next_sync[] to be of size >= (1+ncp)*sizeof(unsigned int) */
void spral_ssids_assemble(const hipStream_t *stream, int nblk, int blkoffset,
      struct assemble_blk_type *blkdata, int ncp,
      struct assemble_cp_type *cpdata, precision_ *children,
      precision_ *parents, unsigned int *gpu_next_sync) {
   /* Create and initialize synchronization objects using a single call:
      next_blk[1]
      sync[ncp]
    */
   CudaSafeCall(
         hipMemsetAsync(gpu_next_sync,0,(1+ncp)*sizeof(unsigned int),*stream)
         );
   /* Note, that we can only have at most 65535 blocks per dimn.
    * For some problems, nblk can exceed this, so we use more than one launch.
    * As the next block we look at is specified by next_blk this works fine.
    */
   dim3 threads(HOGG_ASSEMBLE_NTX, HOGG_ASSEMBLE_NTY);
   for(int i=0; i<nblk; i+=MAX_CUDA_BLOCKS) {
      int blocks = min(MAX_CUDA_BLOCKS, nblk-i);
      assemble
         <HOGG_ASSEMBLE_TX, HOGG_ASSEMBLE_TY,
          HOGG_ASSEMBLE_NTX, HOGG_ASSEMBLE_NTY>
         <<<blocks, threads, 0, *stream>>>
         (&blkdata[blkoffset], cpdata, children, parents, &gpu_next_sync[0],
          &gpu_next_sync[1]);
      CudaCheckError();
   }
}

// Note: modified value lval is passed in via pointer in lndata, not as argument
void spral_ssids_load_nodes( const hipStream_t *stream, int nblocks,
      const struct load_nodes_type *lndata, const long* list,
      const precision_* mval ) {
  for ( int i = 0; i < nblocks; i += MAX_CUDA_BLOCKS ) {
    int nb = min(MAX_CUDA_BLOCKS, nblocks - i);
    cu_load_nodes <<< nb, 128, 0, *stream >>> ( lndata + i, list, mval );
    CudaCheckError();
  }
}

// Note: modified value lval is passed in via pointer in lndata, not as argument
void spral_ssids_load_nodes_sc( const hipStream_t *stream, int nblocks,
      const struct load_nodes_type *lndata, const long* list, const int* rlist,
      const precision_* scale, const precision_* mval ) {
  for ( int i = 0; i < nblocks; i += MAX_CUDA_BLOCKS ) {
    int nb = min(MAX_CUDA_BLOCKS, nblocks - i);
    cu_load_nodes_sc <<< nb, 128, 0, *stream >>> ( lndata + i, list, rlist, scale, mval );
    CudaCheckError();
  }
}

void spral_ssids_max_abs( const hipStream_t *stream, 
      int nb, long n, precision_* u, precision_* buff, precision_* maxabs )
{
  hipMemsetAsync(buff, 0, nb*sizeof(precision_), *stream);
  hipStreamSynchronize(*stream);
  if ( n > 1024*nb )
    cu_max_abs< precision_, 256 ><<< nb, 256, 0, *stream >>>( n, u, buff );
  else
    cu_max_abs< precision_, 32 ><<< nb, 32, 0, *stream >>>( n, u, buff );
  CudaCheckError();
  cu_max_abs< precision_, 1024 ><<< 1, 1024, 0, *stream >>>( nb, buff, maxabs );
  CudaCheckError();
}


} // end extern "C"
