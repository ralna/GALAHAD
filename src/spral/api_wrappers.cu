/* Copyright (c) 2013 Science and Technology Facilities Council (STFC)
 * Authors: Evgueni Ovtchinnikov and Jonathan Hogg
 *
 * This file provides wrappers around functions that are non-trivial to
 * otherwise provide a Fortran interface to using iso_c_binding.
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <ctype.h>
#include <hipblas.h>

// Following wrappers needed as cudaStream_t not interoperable
extern "C" {
   hipError_t spral_cudaStreamCreate(hipStream_t **pStream) {
      *pStream = (hipStream_t *) malloc(sizeof(hipStream_t));
      return hipStreamCreate(*pStream);
   }
   hipError_t spral_cudaStreamDestroy(hipStream_t *stream) {
      hipError_t ret = hipStreamDestroy(*stream);
      free(stream);
      return ret;
   }
   hipError_t spral_cudaMemcpyAsync(void *dst, const void *src, size_t count,
         enum hipMemcpyKind kind, hipStream_t *stream) {
      return hipMemcpyAsync(dst, src, count, kind, *stream);
   }
   hipError_t spral_cudaMemcpy2DAsync(void *dst, size_t dpitch, 
         const void *src, size_t spitch, size_t width, size_t height,
         enum hipMemcpyKind kind, hipStream_t *stream) {
      return hipMemcpy2DAsync(dst, dpitch, src, spitch, width, height, kind, 
         *stream);
   }
   hipError_t spral_cudaMemsetAsync(void *devPtr, int value, size_t count,
         hipStream_t *stream) {
      return hipMemsetAsync(devPtr, value, count, *stream);
   }
   hipError_t spral_cudaStreamSynchronize(hipStream_t *stream) {
      return hipStreamSynchronize(*stream);
   }
}

// Following wrappers needed as cudaEvent_t and cudaStream_t not interoperable
extern "C" {
   hipError_t spral_cudaEventCreateWithFlags(hipEvent_t **event, int flags) {
      *event = (hipEvent_t *) malloc(sizeof(hipEvent_t));
      unsigned int uflags = (unsigned int) flags;
      return hipEventCreateWithFlags(*event, uflags);
   }
   hipError_t spral_cudaEventDestroy(hipEvent_t *event) {
      hipError_t ret = hipEventDestroy(*event);
      free(event);
      return ret;
   }
   hipError_t spral_cudaEventRecord(hipEvent_t *event, hipStream_t *stream) {
      return hipEventRecord(*event, *stream);
   }
   hipError_t spral_cudaEventSynchronize(hipEvent_t *event) {
      return hipEventSynchronize(*event);
   }
}

// Following wrappers needed as cublasHandle_T not interoperable
extern "C" {
   hipblasStatus_t spral_cublasCreate(hipblasHandle_t **const handle)
   {
      *handle = (hipblasHandle_t*)malloc(sizeof(hipblasHandle_t));
      return hipblasCreate(*handle);
   }
   hipblasStatus_t spral_cublasDestroy(hipblasHandle_t *const handle)
   {
      const hipblasStatus_t error = hipblasDestroy(*handle);
      free(handle);
      return error;
   }
   hipblasStatus_t spral_cublasDgemm(hipblasHandle_t *const handle, 
                                    const char *const transa,
                                    const char *const transb, 
                                    const int *const m, 
                                    const int *const n, 
                                    const int *const k,
                                    const double *const alpha, 
                                    const double *const devPtrA, 
                                    const int *const lda,
                                    const double *const devPtrB, 
                                    const int *const ldb, 
                                    const double *const beta,
                                    double *const devPtrC, 
                                    const int *const ldc)
  {
      hipblasOperation_t tA, tB;
      if (toupper(*transa) == 'N') 
        tA = HIPBLAS_OP_N;
      else
        tA = HIPBLAS_OP_T;
      if (toupper(*transb) == 'N') 
        tB = HIPBLAS_OP_N;
      else
         tB = HIPBLAS_OP_T;
      return hipblasDgemm(*handle, tA, tB, *m, *n, *k, alpha, devPtrA, *lda, 
                         devPtrB, *ldb, beta, devPtrC, *ldc);
   }
   hipblasStatus_t spral_cublasSgemm(hipblasHandle_t *const handle, 
                                    const char *const transa,
                                    const char *const transb, 
                                    const int *const m, 
                                    const int *const n, 
                                    const int *const k,
                                    const float *const alpha, 
                                    const float *const devPtrA, 
                                    const int *const lda,
                                    const float *const devPtrB, 
                                    const int *const ldb, 
                                    const float *const beta,
                                    float *const devPtrC, 
                                    const int *const ldc)
  {
      hipblasOperation_t tA, tB;
      if (toupper(*transa) == 'N') 
        tA = HIPBLAS_OP_N;
      else
        tA = HIPBLAS_OP_T;
      if (toupper(*transb) == 'N') 
        tB = HIPBLAS_OP_N;
      else
         tB = HIPBLAS_OP_T;
      return hipblasSgemm(*handle, tA, tB, *m, *n, *k, alpha, devPtrA, *lda, 
                         devPtrB, *ldb, beta, devPtrC, *ldc);
   }
   hipblasStatus_t spral_cublasSetStream(hipblasHandle_t *handle,
         hipStream_t *streamId) {
      return hipblasSetStream(*handle, *streamId);
   }
}

/*
 * Exceptionally useful non-CUDA API functions
 */

// Used to provide pointer arithmetic in Fortran
extern "C"
void *spral_c_ptr_plus(void *const base, const size_t sz)
{
  return (void*)(((char*)base) + sz);
}

// Allow pretty printing of a C pointer in Fortran
extern "C"
void spral_c_print_ptr(void *const ptr)
{
  (void)printf("ptr = %p\n", ptr);
}
